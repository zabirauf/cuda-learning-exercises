#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define CHECK_CUDA_ERROR() do { \
    hipError_t error = hipGetLastError(); \
    if (error != hipSuccess) { \
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error)); \
        exit(-1); \
    } \
} while(0)

__global__ void add_kernel(float *a, float *b, float *c, uint n) {
    uint idx = threadIdx.x;

    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    // Select the GPU you want to use (0 for the first GPU, 1 for the second, etc.)
    // int deviceId = 0; // Change this to 1 if you want to use the second GPU
    // cudaError_t error = cudaSetDevice(deviceId);
    // if (error != cudaSuccess) {
    //     fprintf(stderr, "cudaSetDevice failed! Error: %s\n", cudaGetErrorString(error));
    //     return 1;
    // }

    float *a_h = (float *)malloc(10 * sizeof(float));
    a_h[0] = 1.0;
    a_h[1] = 2.0;
    a_h[2] = 3.0;
    a_h[3] = 4.0;
    a_h[4] = 5.0;
    a_h[5] = 6.0;
    a_h[6] = 7.0;
    a_h[7] = 8.0;
    a_h[8] = 9.0;
    a_h[9] = 10.0;

    float *b_h = (float *)malloc(10 * sizeof(float));
    b_h[0] = 10.0;
    b_h[1] = 11.0;
    b_h[2] = 12.0;
    b_h[3] = 13.0;
    b_h[4] = 14.0;
    b_h[5] = 15.0;
    b_h[6] = 16.0;
    b_h[7] = 17.0;
    b_h[8] = 18.0;
    b_h[9] = 19.0;



    float *a_d, *b_d, *c_d;

    hipMalloc((void**)&a_d, 10 * sizeof(float));
    hipMalloc((void**)&b_d, 10 * sizeof(float));
    hipMalloc((void**)&c_d, 10 * sizeof(float));

    hipMemcpy(a_d, a_h, 10 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, 10 * sizeof(float), hipMemcpyHostToDevice);

    add_kernel<<<1, 10>>>(a_d, b_d, c_d, 10);
    
    CHECK_CUDA_ERROR();

    float *c_h = (float *)malloc(10 * sizeof(float));
    hipMemcpy(c_h, c_d, 10 * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++) {
        printf("%f\n", c_h[i]);
    }

    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);

    free(a_h);
    free(b_h);
    free(c_h);

    return 0;
}